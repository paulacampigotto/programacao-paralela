
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
//__constant__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 2;
	dim3 dimBlock (16,16);
	dim3 dimGrid(32,32);
	hello<<<dimGrid,dimBlock>>>();
	hipDeviceSynchronize();

	return 0;
}
