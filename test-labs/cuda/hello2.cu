
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%c dimGrid(%d) dimBlock\n", STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 2;
	dim3 dimBlock (16,16);
	dim3 dimGrid(32,32);
	hello<<<dimGrid,dimBlock>>>();
	hipDeviceSynchronize();

	return 0;
}
